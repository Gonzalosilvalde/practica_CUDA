#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define N 50

// Función kernel para inicializar la matriz con valores aleatorios
__global__ void initializeMatrix(float *matrix, unsigned int seed) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int index = i * N + j;
    
    // Inicializa el generador de números aleatorios para cada hilo
    hiprandState_t state;
    hiprand_init(seed, index, 0, &state);
    
    // Asegurarse de no exceder los límites de la matriz
    if (i < N && j < N) {
        matrix[index] = hiprand_uniform(&state) * 100; // Asigna un valor aleatorio entre 0 y 99
    }
}

__global__ void sumMatrix(float *matrix, float *result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int index = i * N + j;

    if (i < N && j < N) {
        atomicAdd(result, matrix[index]);
    }
}

int main() {
    float *matrix;
    size_t size = N * N * sizeof(float);
    
    // Aloja memoria en el dispositivo CUDA para la matriz
    hipMalloc(&matrix, size);
    
    // Define las dimensiones del grid y del bloque
    dim3 blockSize(16, 16); // Bloque de 16x16 hilos
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    
    // Define una semilla para los números aleatorios
    unsigned int seed = time(NULL);
    
    // Llama al kernel para inicializar la matriz con valores aleatorios
    initializeMatrix<<<gridSize, blockSize>>>(matrix, seed);
    
    // Espera a que todos los threads finalicen
    hipDeviceSynchronize();

    // Aloja memoria en el dispositivo CUDA para el resultado
    float *d_result;
    hipMalloc(&d_result, sizeof(float));
    
    // Inicializa el resultado en 0 en el dispositivo CUDA
    hipMemset(d_result, 0, sizeof(float));
    
    // Llama al kernel para sumar la matriz y almacenar el resultado en d_result
    sumMatrix<<<gridSize, blockSize>>>(matrix, d_result);
    
    // Copia el resultado desde el dispositivo al host
    float h_result;
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    
    // Muestra el resultado
    printf("Suma de la matriz: %f\n", h_result);
    
    // Libera la memoria de la matriz y el resultado en el dispositivo CUDA
    hipFree(matrix);
    hipFree(d_result);
    
    return 0;
}
